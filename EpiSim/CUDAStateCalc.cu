#include "hip/hip_runtime.h"
#pragma once
#include <vector>
#include <boost/random.hpp>
#include <boost/random/random_device.hpp>
std::vector<std::vector<int>> finished;
boost::random::mt11213b  generator;
boost::random::mt11213b  generator_x;
boost::random::mt11213b  generator_y;
boost::random::random_device seeder;

#ifdef __NVCC__
#define THREAD_GROUP_SIZE 256

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include ""
#include <hiprand/hiprand_kernel.h>
#include <math.h>

/*bool infection(float infectionProbability) {
	int number = 0;
	number = generator() % 100;
	return number < infectionProbability;
}

vector<vector<signed int>> generateCoords(int64_t amount, int area[2], int purpose) {
	vector<vector<int>> coords;
	int modulo_x = 3;
	int modulo_y = 3;
	int minus = 1;
	if (purpose != 0) {
		modulo_x = area[0];
		modulo_y = area[1];
		minus = 0;
	}
	for (int i = 0; i < amount; i++) {
		vector<int> temp;
		temp.push_back((generator_x() % modulo_x) - minus);
		temp.push_back((generator_y() % modulo_y) - minus);
		coords.push_back(temp);
	}
	return coords;
}

void threaded(vector<vector<int>>* EntireLastState, int start, int end, int id, int area[2], vector<int> infectedPeople, int infectionRadius, float infectionProbability) {
	logger::log(0, 0, "Thread " + to_string(id) + " reporting for duty!");
	vector<vector<int>> offsets;
	vector<vector<int>> result;
	int size[2] = { 1,1 };
	offsets = generateCoords(end - start, size, 0);

	for (int i = 0; i != (end - start); i++) {
		vector<int> temppair;
		temppair.push_back(EntireLastState->at(i + start)[0] + offsets[i][0]);
		if (temppair[0] > area[0]) {
			temppair[0] = area[0] - 1;
		}
		if (temppair[0] < 0) {
			temppair[0] = 0;
		}
		temppair.push_back(EntireLastState->at(i + start)[1] + offsets[i][1]);
		if (temppair[1] > area[1]) {
			temppair[1] = area[1] - 1;
		}
		if (temppair[1] < 0) {
			temppair[1] = 0;
		}
		int infected = EntireLastState->at(i + start)[2];
		if (!(std::find(infectedPeople.begin(), infectedPeople.end(), i) != infectedPeople.end())) {
			for (int j = 0; j < infectedPeople.size(); j++) {
				auto distance = std::sqrt(pow((EntireLastState->at(infectedPeople[j])[0] - temppair[0]), 2) + pow((EntireLastState->at(infectedPeople[j])[1] - temppair[1]), 2));
				logger::log(5, 0, to_string(distance));
				if (distance < infectionRadius) {
					if (infection(infectionProbability)) {
						infected = 1;
						infectedPeople.push_back(i);
					}
				}
			}
		}
		temppair.push_back(infected);

		//logger::log(0, 0, "Person " + to_string(i + start) + " has been calculated");
		finished[i + start] = temppair;
	}
	logger::log(0, 0, "Thread " + to_string(id) + " has finished");
}

void calculateNextState(std::vector<std::vector<int>>* lastState, int area[2], std::vector<int> infectedPeople, int infectionRadius, float infectionProbability, int threads) {
	std::vector<int> offsets;
	for (int i = 0; i < threads + 1; i++) {
		offsets.push_back(i * (lastState->size() / threads));
	}
	offsets[threads] += (lastState->size() % threads);
	//boost::thread worker(threaded, lastState, offsets[0], offsets[1]-1, 0, area, infectedPeople, infectionRadius, infectionProbability);
	boost::thread_group threadgroup;
	for (int i = 0; i < threads; i++) {
		threadgroup.create_thread(boost::bind(threaded, lastState, offsets[i], offsets[i + 1], i, area, infectedPeople, infectionRadius, infectionProbability));
	}
	threadgroup.join_all();
}
*/


const int    DEFAULT_RAND_N = 2400000;
const unsigned int DEFAULT_SEED = 777;

__global__ void cuda_kernel(int amount, int *x, int *y, int *i_x, int *i_y, bool *s, int infectedRadius, float infectionProbability, int *area, hiprandState *state) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < amount) {
		float out;
		out = hiprand_uniform(state) * 2 - 1;
		x[i] += (int)out;
		out = hiprand_uniform(state) * 2 - 1;
		y[i] += (int)out;
		for (int j = 0; i < sizeof(i_x)/sizeof(int); j++) {
			float distance = (float)sqrt(pow((float)(i_x[j] - x[i]), (float)2) + pow((float)(i_y[j] - y[i]), (float)2));
			if (distance < infectedRadius) {
				out = hiprand_uniform(state)*100.0f;

				if (((int)out) < infectionProbability) {
					s[i] = true;
				}
			}
		}
	}
}

void CUDA_calculateNextState(std::vector<std::vector<int>>* lastState, int area[2], std::vector<int> infectedPeople, int infectionRadius, float infectionProbability, int threads) {
	int amount = lastState->size();
	int infected_size = infectedPeople.size();
	int* x, * y, * i_x, *i_y, * d_x, * d_y, * di_x, * di_y, *d_area;
	bool* s, * d_s;
	hiprandState* d_state;

	////// Allocation
	//Host Mallocs
	x = (int*)malloc(amount * sizeof(int));
	y = (int*)malloc(amount * sizeof(int));
	i_x = (int*)malloc(infected_size * sizeof(int));
	i_y = (int*)malloc(infected_size * sizeof(int));
	s = (bool*)malloc(amount * sizeof(bool));

	//CUDA Mallocs
	hipMalloc(&d_state, sizeof(hiprandState));
	hipMalloc(&d_x, amount * sizeof(int));
	hipMalloc(&d_y, amount * sizeof(int));
	hipMalloc(&di_x, infected_size * sizeof(int));
	hipMalloc(&di_y, infected_size * sizeof(int));
	hipMalloc(&d_s, amount * sizeof(bool));
	hipMalloc(&d_area, 2 * sizeof(int));
	for (int i = 0; i < amount; i++) {
		x[i] = lastState->at(i)[0];
		y[i] = lastState->at(i)[1];
		s[i] = false;
	}

	for (int i = 0; i < infected_size; i++) {
		i_x[i] = lastState->at(infectedPeople.at(i))[0];
		i_y[i] = lastState->at(infectedPeople.at(i))[1];
	}

	////// CUDA Memcpys
	hipMemcpy(d_x, x, amount * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, amount * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(di_x, x, amount * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(di_y, y, amount * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_s, s, amount * sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(d_area, area, 2 * sizeof(int), hipMemcpyHostToDevice);

	cuda_kernel<<<(amount + (THREAD_GROUP_SIZE - 1)) / THREAD_GROUP_SIZE, THREAD_GROUP_SIZE>>>(amount, d_x, d_y, di_x, di_y, d_s, infectionRadius, infectionRadius, d_area, d_state);
	hipDeviceSynchronize();
	hipMemcpy(s, d_s, amount * sizeof(bool), hipMemcpyDeviceToHost);
	hipMemcpy(x, d_x, amount * sizeof(bool), hipMemcpyDeviceToHost);
	hipMemcpy(y, d_y, amount * sizeof(bool), hipMemcpyDeviceToHost);

	hipFree(d_x);
	hipFree(d_y);
	hipFree(di_x);
	hipFree(di_y);
	hipFree(d_s);
	hipFree(d_area);

	for (int i = 0; i < sizeof(x)/sizeof(int); i++) {
		std::vector<int> tempvec;
		tempvec.push_back(x[i]);
		tempvec.push_back(y[i]);
		if (s[i]) {
			tempvec.push_back(1);
		}
		else {
			tempvec.push_back(0);
		}
	}

	free(x);
	free(y);
	free(i_x);
	free(i_y);
	free(s);
}
#endif

void calculateNextState(std::vector<std::vector<int>>* lastState, int area[2], std::vector<int> infectedPeople, int infectionRadius, float infectionProbability, int threads) {
#ifdef __NVCC__
	CUDA_calculateNextState(lastState, area, infectedPeople, infectionRadius, infectionProbability, threads);
#endif
}